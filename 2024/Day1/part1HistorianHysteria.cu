#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void calculateDistances(int *left, int *right, int *distances,
                                   int n) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    distances[idx] = abs(left[idx] - right[idx]);
  }
}

void sort(int *arr, int n) {
  for (int i = 0; i < n - 1; i++) {
    for (int j = 0; j < n - i - 1; j++) {
      if (arr[j] > arr[j + 1]) {
        int temp = arr[j];
        arr[j] = arr[j + 1];
        arr[j + 1] = temp;
      }
    }
  }
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

int main() {
  FILE *file = fopen("input.txt", "r");
  if (file == NULL) {
    printf("Error opening file\n");
    return 1;
  }

  int left[1000], right[1000];
  int n = 0;

  while (fscanf(file, "%d %d", &left[n], &right[n]) != EOF) {
    n++;
  }
  fclose(file);

  // Debugging: Print the input arrays
  printf("Left array:\n");
  for (int i = 0; i < n; i++) {
    printf("%d ", left[i]);
  }
  printf("\nRight array:\n");
  for (int i = 0; i < n; i++) {
    printf("%d ", right[i]);
  }
  printf("\n");

  sort(left, n);
  sort(right, n);

  // Debugging: Print the sorted arrays
  printf("Sorted left array:\n");
  for (int i = 0; i < n; i++) {
    printf("%d ", left[i]);
  }
  printf("\nSorted right array:\n");
  for (int i = 0; i < n; i++) {
    printf("%d ", right[i]);
  }
  printf("\n");

  int *d_left, *d_right, *d_distances;
  int *distances = (int *)malloc(n * sizeof(int));

  checkCudaError(hipMalloc((void **)&d_left, n * sizeof(int)),
                 "Allocating d_left");
  checkCudaError(hipMalloc((void **)&d_right, n * sizeof(int)),
                 "Allocating d_right");
  checkCudaError(hipMalloc((void **)&d_distances, n * sizeof(int)),
                 "Allocating d_distances");

  checkCudaError(
      hipMemcpy(d_left, left, n * sizeof(int), hipMemcpyHostToDevice),
      "Copying left to d_left");
  checkCudaError(
      hipMemcpy(d_right, right, n * sizeof(int), hipMemcpyHostToDevice),
      "Copying right to d_right");

  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  calculateDistances<<<numBlocks, blockSize>>>(d_left, d_right, d_distances, n);
  checkCudaError(hipGetLastError(), "Kernel launch");
  checkCudaError(hipDeviceSynchronize(), "Kernel execution");

  checkCudaError(hipMemcpy(distances, d_distances, n * sizeof(int),
                            hipMemcpyDeviceToHost),
                 "Copying distances to host");

  // Debugging: Print the distances array
  printf("Distances array:\n");
  for (int i = 0; i < n; i++) {
    printf("%d ", distances[i]);
  }
  printf("\n");

  int totalDistance = 0;
  for (int i = 0; i < n; i++) {
    totalDistance += distances[i];
  }

  printf("Total Distance: %d\n", totalDistance);

  hipFree(d_left);
  hipFree(d_right);
  hipFree(d_distances);
  free(distances);

  return 0;
}
